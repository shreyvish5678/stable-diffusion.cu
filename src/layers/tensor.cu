#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include "kernels.h"
#include <fstream>
#include <string>
#include "tensor.h"

// Default Constructor
Tensor::Tensor() {
    data_cpu = NULL;
    data_gpu = NULL;
    dims = NULL;
    ndims = 0;
    size = 0;
}
// Constructor: Create a tensor with the given properties
Tensor::Tensor(int* dims, int ndims) {
    this->ndims = ndims;
    this->dims = (int*)malloc(ndims * sizeof(int));
    size = 1;
    for (int i = 0; i < ndims; i++) {
        this->dims[i] = dims[i];
        size *= dims[i];
    }
    allocate_cpu();
    allocate_gpu();
}
// Allocate memory on CPU
void Tensor::allocate_cpu() {
    hipError_t err = hipHostMalloc((void**)&data_cpu, size * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Error allocating memory on CPU" << std::endl;
        exit(1);
    }
}
// Allocate memory on GPU
void Tensor::allocate_gpu() {
    hipError_t err = hipMalloc((void**)&data_gpu, size * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Error allocating memory on GPU" << std::endl;
        exit(1);
    }
}
// Destructor
void Tensor::free_memory() {
    hipHostFree(data_cpu);
    hipFree(data_gpu);
}

void Tensor::init_zero() {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    init_zero_kernel<<<num_blocks, block_size>>>(data_gpu, size);
    hipDeviceSynchronize();
    CHECK_ERROR();
    hipMemcpy(data_cpu, data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
}

void Tensor::init_one() {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    init_one_kernel<<<num_blocks, block_size>>>(data_gpu, size);
    hipDeviceSynchronize();
    CHECK_ERROR();
    hipMemcpy(data_cpu, data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
}

void Tensor::init_rand() {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    unsigned long long seed = rand();
    init_rand_kernel<<<num_blocks, block_size>>>(data_gpu, size, seed);
    hipDeviceSynchronize();
    CHECK_ERROR();
    hipMemcpy(data_cpu, data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
}

float Tensor::at(int i) {
    if (ndims != 1) {
        throw std::runtime_error("Wrong number of indices");
    }
    return data_cpu[i];
}

// Following methods are used to access elements of the tensor
float Tensor::at(int i, int j) {
    if (ndims != 2) {
        throw std::runtime_error("Wrong number of indices");
    }
    return data_cpu[i * dims[1] + j];
}

float Tensor::at(int i, int j, int k) {
    if (ndims != 3) {
        throw std::runtime_error("Wrong number of indices");
    }
    return data_cpu[i * dims[1] * dims[2] + j * dims[2] + k];
}

float Tensor::at(int* indices) {
    int index = 0;
    for (int i = 0; i < ndims; i++) {
        index = index * dims[i] + indices[i];
    }
    return data_cpu[index];
}

// Following methods are used to print the whole tensor
void Tensor::print() {
    print_recursive(data_cpu, 0, 0);
    std::cout << std::endl;
}

// Method to perform general operations on tensors
Tensor Tensor::operation(Tensor& other, int operation) {
    Tensor result = Tensor(dims, ndims);
    if (size == other.size && ndims == other.ndims) {
        for (int i = 0; i < ndims; i++) {
            assert(dims[i] == other.dims[i]);
        }
        int block_size = 256;
        int num_blocks = (size + block_size - 1) / block_size;
        ops_kernel<<<num_blocks, block_size>>>(result.data_gpu, data_gpu, other.data_gpu, size, operation);
        hipDeviceSynchronize();
        CHECK_ERROR();
        hipMemcpy(result.data_cpu, result.data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
    if (ndims == 2 && other.ndims == 1) {
        assert(dims[1] == other.dims[0]);
        dim3 block_size(16, 16);
        dim3 num_blocks((dims[1] + block_size.x - 1) / block_size.x, (dims[0] + block_size.y - 1) / block_size.y);
        ops_bias_kernel<<<num_blocks, block_size>>>(result.data_gpu, data_gpu, other.data_gpu, dims, other.dims, dims[0], dims[1], other.dims[0], operation);
        hipDeviceSynchronize();
        CHECK_ERROR();
        hipMemcpy(result.data_cpu, result.data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
    else if (ndims == 3 && other.ndims == 1) {
        assert(dims[2] == other.dims[0]);
        dim3 block_size(16, 16);
        dim3 num_blocks((dims[2] + block_size.x - 1) / block_size.x, (dims[1] + block_size.y - 1) / block_size.y, dims[0]);
        ops_bias_3d_kernel<<<num_blocks, block_size>>>(result.data_gpu, data_gpu, other.data_gpu, dims, other.dims, dims[0], dims[1], dims[2], other.dims[0], operation);
        hipDeviceSynchronize();
        CHECK_ERROR();
        hipMemcpy(result.data_cpu, result.data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
    else if (ndims == 3 && other.ndims == 2) {
        if (dims[0] == other.dims[0]) {
            assert(dims[1] == other.dims[1]);
            dim3 block_size(8, 8, 8);
            dim3 num_blocks((dims[1] + block_size.x - 1) / block_size.x, (dims[0] + block_size.y - 1) / block_size.y, (dims[2] + block_size.z - 1) / block_size.z);
            ops_channel_kernel<<<num_blocks, block_size>>>(result.data_gpu, data_gpu, other.data_gpu, dims[0], dims[1], dims[2], operation);
            hipDeviceSynchronize();
            CHECK_ERROR();
            hipMemcpy(result.data_cpu, result.data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
            return result;
        }
        else {
            assert(dims[1] == other.dims[0]);
            assert(dims[2] == other.dims[1]);
            dim3 block_size(8, 8, 8);
            dim3 num_blocks((dims[1] + block_size.x - 1) / block_size.x, (dims[0] + block_size.y - 1) / block_size.y, (dims[2] + block_size.z - 1) / block_size.z);
            ops_batch_kernel<<<num_blocks, block_size>>>(result.data_gpu, data_gpu, other.data_gpu, dims[0], dims[1], dims[2], operation);
            hipDeviceSynchronize();
            CHECK_ERROR();
            hipMemcpy(result.data_cpu, result.data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
            return result;
        }
    }
    else {
        std::cerr << "Invalid dimensions for addition" << std::endl;
        exit(1);
    }
}

Tensor Tensor::operation(float scalar, int operation) {
    Tensor result = Tensor(dims, ndims);
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    ops_scalar_kernel<<<num_blocks, block_size>>>(result.data_gpu, data_gpu, scalar, size, operation);
    hipDeviceSynchronize();
    CHECK_ERROR();
    hipMemcpy(result.data_cpu, result.data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
    return result;
}

// Overloaded operators
Tensor Tensor::operator+(Tensor& other) {
    return operation(other, 0);
}

Tensor Tensor::operator-(Tensor& other) {
    return operation(other, 1);
}

Tensor Tensor::operator*(Tensor& other) {
    return operation(other, 2);
}

Tensor Tensor::operator/(Tensor& other) {
    return operation(other, 3);
}

Tensor Tensor::operator+(float scalar) {
    return operation(scalar, 0);
}

Tensor Tensor::operator-(float scalar) {
    return operation(scalar, 1);
}

Tensor Tensor::operator*(float scalar) {
    return operation(scalar, 2);
}

Tensor Tensor::operator/(float scalar) {
    return operation(scalar, 3);
}

// Static method to perform matrix multiplication
Tensor Tensor::matmul(Tensor& a, Tensor& b) {
    if (a.ndims == 2 && b.ndims == 2) {
        assert(a.dims[1] == b.dims[0]);

        int result_dims[] = {a.dims[0], b.dims[1]};
        Tensor result = Tensor(result_dims, 2);

        dim3 block_size(16, 16);
        dim3 num_blocks((result_dims[1] + block_size.x - 1) / block_size.x, (result_dims[0] + block_size.y - 1) / block_size.y);
        matmul_kernel<<<num_blocks, block_size>>>(result.data_gpu, a.data_gpu, b.data_gpu, a.dims[0], b.dims[0], a.dims[1], b.dims[1]);
        hipDeviceSynchronize();
        CHECK_ERROR();

        hipMemcpy(result.data_cpu, result.data_gpu, result.size * sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
    else if (a.ndims == 3 && b.ndims == 2) {
        assert (a.dims[2] == b.dims[0]);

        int result_dims[] = {a.dims[0], a.dims[1], b.dims[1]};
        Tensor result = Tensor(result_dims, 3);

        dim3 block_size(16, 16);
        dim3 num_blocks((result_dims[2] + block_size.x - 1) / block_size.x, (result_dims[1] + block_size.y - 1) / block_size.y, result_dims[0]);

        matmul_3d_kernel<<<num_blocks, block_size>>>(result.data_gpu, a.data_gpu, b.data_gpu, a.dims[0], a.dims[1], b.dims[0], a.dims[2], b.dims[1]);
        hipDeviceSynchronize();
        CHECK_ERROR();

        hipMemcpy(result.data_cpu, result.data_gpu, result.size * sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
    else if (a.ndims == 4 && b.ndims == 4) {
        assert(a.dims[0] == b.dims[0]);
        assert (a.dims[1] == b.dims[1]);
        assert (a.dims[3] == b.dims[2]);
        int result_dims[] = {a.dims[0], a.dims[1], a.dims[2], b.dims[3]};
        Tensor result = Tensor(result_dims, 4);
        dim3 block_size(16, 16);
        dim3 num_blocks((result_dims[3] + block_size.x - 1) / block_size.x, 
                        (result_dims[2] + block_size.y - 1) / block_size.y, 
                        result_dims[0] * result_dims[1]);

        matmul_4d_kernel<<<num_blocks, block_size>>>(
            result.data_gpu, a.data_gpu, b.data_gpu, 
            a.dims[0], a.dims[1], a.dims[2], a.dims[3], b.dims[3]
        );

        hipDeviceSynchronize();
        CHECK_ERROR();

        hipMemcpy(result.data_cpu, result.data_gpu, result.size * sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
    else {
        std::cout << a.ndims << " " << b.ndims << std::endl;
        std::cerr << "Invalid dimensions for matrix multiplication" << std::endl;
        exit(1);
    }
}

// Reshape the tensor
Tensor Tensor::reshape(int* new_dims, int new_ndims) {
    int new_size = 1;
    for (int i = 0; i < new_ndims; i++) {
        new_size *= new_dims[i];
    }
    assert(new_size == size);

    Tensor result = Tensor(new_dims, new_ndims);
    hipMemcpy(result.data_gpu, data_gpu, size * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(result.data_cpu, result.data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
    return result;
}

// Transpose the tensor
Tensor Tensor::transpose(int dim1, int dim2) {
    assert (ndims == 4);
    int new_dims[] = {dims[0], dims[1], dims[2], dims[3]};
    std::swap(new_dims[dim1], new_dims[dim2]);
    Tensor result = Tensor(new_dims, ndims);
    int block_size = 256;
    int grid_size = (size + block_size - 1) / block_size;
    transpose_kernel<<<grid_size, block_size>>>(result.data_gpu, data_gpu, dims[0], dims[1], dims[2], dims[3], dim1, dim2, size);
    hipDeviceSynchronize();
    CHECK_ERROR();
    hipMemcpy(result.data_cpu, result.data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
    return result;
}

// Softmax function 
Tensor Tensor::softmax() {
    assert (ndims == 4);
    Tensor result = Tensor(dims, ndims);
    dim3 block_size(dims[0], dims[1], dims[2]);
    dim3 threads(dims[3], 1, 1);
    hipMemcpy(data_gpu, data_cpu, size * sizeof(float), hipMemcpyHostToDevice);
    softmax_kernel<<<block_size, threads>>>(result.data_gpu, data_gpu, dims[0], dims[1], dims[2], dims[3]);
    hipDeviceSynchronize();
    CHECK_ERROR();
    hipMemcpy(result.data_cpu, result.data_gpu, size * sizeof(float), hipMemcpyDeviceToHost);
    return result;
}

// Mean function
Tensor Tensor::mean(Tensor& input) {
    assert (input.ndims == 4 || input.ndims == 3);
    if (input.ndims == 4) {
        int new_dims[] = {input.dims[0], input.dims[1], input.dims[2]}; 
        Tensor result = Tensor(new_dims, 3);
        dim3 block_size(16, 16);
        dim3 num_blocks((new_dims[1] + block_size.x - 1) / block_size.x, (new_dims[0] + block_size.y - 1) / block_size.y);
        mean_kernel<<<num_blocks, block_size>>>(result.data_gpu, input.data_gpu, input.dims[0] * input.dims[1], input.dims[2], input.dims[3]);
        hipDeviceSynchronize();
        CHECK_ERROR();
        hipMemcpy(result.data_cpu, result.data_gpu, result.size * sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
    else {
        int new_dims[] = {input.dims[0], input.dims[1]};
        Tensor result = Tensor(new_dims, 2);
        dim3 block_size(16, 16);
        dim3 num_blocks((new_dims[1] + block_size.x - 1) / block_size.x, (new_dims[0] + block_size.y - 1) / block_size.y);
        mean_kernel<<<num_blocks, block_size>>>(result.data_gpu, input.data_gpu, input.dims[0], input.dims[1], input.dims[2]);
        hipDeviceSynchronize();
        CHECK_ERROR();
        hipMemcpy(result.data_cpu, result.data_gpu, result.size * sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
}

// Variance function
Tensor Tensor::variance(Tensor& input, Tensor& mean) {
    assert (input.ndims == 4 || input.ndims == 3);
    if (input.ndims == 4) {
        int new_dims[] = {input.dims[0], input.dims[1], input.dims[2]}; 
        Tensor result = Tensor(new_dims, 3);
        dim3 block_size(16, 16);
        dim3 num_blocks((new_dims[1] + block_size.x - 1) / block_size.x, (new_dims[0] + block_size.y - 1) / block_size.y);
        variance_kernel<<<num_blocks, block_size>>>(result.data_gpu, input.data_gpu, mean.data_gpu, input.dims[0] * input.dims[1], input.dims[2], input.dims[3]);
        hipDeviceSynchronize();
        CHECK_ERROR();
        hipMemcpy(result.data_cpu, result.data_gpu, result.size * sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
    else {
        int new_dims[] = {input.dims[0], input.dims[1]};
        Tensor result = Tensor(new_dims, 2);
        dim3 block_size(16, 16);
        dim3 num_blocks((new_dims[1] + block_size.x - 1) / block_size.x, (new_dims[0] + block_size.y - 1) / block_size.y);
        variance_kernel<<<num_blocks, block_size>>>(result.data_gpu, input.data_gpu, mean.data_gpu, input.dims[0], input.dims[1], input.dims[2]);
        hipDeviceSynchronize();
        CHECK_ERROR();
        hipMemcpy(result.data_cpu, result.data_gpu, result.size * sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
}

// Inverse square root function
Tensor Tensor::sqrt(Tensor& input) {
    Tensor result = Tensor(input.dims, input.ndims);
    int block_size = 256;
    int num_blocks = (input.size + block_size - 1) / block_size;
    sqrt_kernel<<<num_blocks, block_size>>>(result.data_gpu, input.data_gpu, input.size);
    hipDeviceSynchronize();
    CHECK_ERROR();
    hipMemcpy(result.data_cpu, result.data_gpu, result.size * sizeof(float), hipMemcpyDeviceToHost);
    return result;
}

// QuickGELU function
Tensor Tensor::gelu(Tensor& input) {
    Tensor result = Tensor(input.dims, input.ndims);
    int block_size = 256;
    int num_blocks = (input.size + block_size - 1) / block_size;
    gelu_kernel<<<num_blocks, block_size>>>(result.data_gpu, input.data_gpu, input.size);
    hipDeviceSynchronize();
    CHECK_ERROR();
    hipMemcpy(result.data_cpu, result.data_gpu, result.size * sizeof(float), hipMemcpyDeviceToHost);
    return result;
}

// Sigmoid Linear Unit function
Tensor Tensor::silu(Tensor& input) {
    Tensor result = Tensor(input.dims, input.ndims);
    int block_size = 256;
    int num_blocks = (input.size + block_size - 1) / block_size;
    silu_kernel<<<num_blocks, block_size>>>(result.data_gpu, input.data_gpu, input.size);
    hipDeviceSynchronize();
    CHECK_ERROR();
    hipMemcpy(result.data_cpu, result.data_gpu, result.size * sizeof(float), hipMemcpyDeviceToHost);
    return result;
}

// Save the tensor to a file
void Tensor::save(const std::string& filename) {
    std::ofstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error opening file" << std::endl;
        exit(1);
    }
    int total_size = 1;
    for (int i = 0; i < ndims; ++i) {
        total_size *= dims[i];
    }
    file.write(reinterpret_cast<char*>(data_cpu), total_size * sizeof(float));

    file.close();
}

// Load the tensor from a file
void Tensor::load(const std::string& filename) {
    std::ifstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error opening file" << std::endl;
        exit(1);
    }
    int total_size = 1;
    for (int i = 0; i < ndims; ++i) {
        total_size *= dims[i];
    }
    file.read(reinterpret_cast<char*>(data_cpu), total_size * sizeof(float));

    file.close();

    hipMemcpy(data_gpu, data_cpu, size * sizeof(float), hipMemcpyHostToDevice);
}

void Tensor::print_recursive(float* data, int dim_idx, int offset) {
    if (dim_idx == ndims) {
        std::cout << data[offset] << " ";
        return;
    }

    int stride = 1;
    for (int i = dim_idx + 1; i < ndims; i++) {
        stride *= dims[i];
    }

    std::cout << "[";
    for (int i = 0; i < dims[dim_idx]; i++) {
        print_recursive(data, dim_idx + 1, offset + i * stride);
        if (i != dims[dim_idx] - 1) std::cout << " ";
    }
    std::cout << "]";
}