#include <iostream>
#include "src/clip.h"

int main() {
    int tokens[2][77];
    for (int i = 0; i < 2; i++) {
        for (int j = 0; j < 77; j++) {
            tokens[i][j] = rand() % 100;   
        }
    }
    CLIP clip = CLIP();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    Tensor output = clip.forward(&tokens[0][0], 2);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time: " << milliseconds << "ms" << std::endl;
    std::cout << "Output: " << output.data_cpu[0] << std::endl;
    return 0;
}