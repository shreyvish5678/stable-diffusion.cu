#include "hip/hip_runtime.h"
#include "kernels.h"
#include "tensor.h"
#include "linear.h"
#include "attention.h"

SelfAttention::SelfAttention() {
    heads = 0;
    d_embed = 0;
    d_head = 0;
}

SelfAttention::SelfAttention(int heads, int d_embed, bool proj_bias, bool out_bias) {
    q_proj = Linear(d_embed, d_embed, proj_bias);
    k_proj = Linear(d_embed, d_embed, proj_bias);
    v_proj = Linear(d_embed, d_embed, proj_bias);
    out_proj = Linear(d_embed, d_embed, out_bias);
    this->heads = heads;
    this->d_embed = d_embed;
    this->d_head = d_embed / heads;
}

Tensor SelfAttention::forward(Tensor& input, bool mask) {
    int batch_size = input.dims[0];
    int seq_len = input.dims[1];
    int interim_dims[] = {batch_size, seq_len, heads, d_head};
    Tensor q = q_proj.forward(input).reshape(interim_dims, 4).transpose(1, 2);
    Tensor k = k_proj.forward(input).reshape(interim_dims, 4).transpose(1, 2).transpose(2, 3);
    Tensor v = v_proj.forward(input).reshape(interim_dims, 4).transpose(1, 2);
    Tensor weight = Tensor::matmul(q, k);
    if (mask) { 
        dim3 block_size(16, 16);
        dim3 grid_size((seq_len + block_size.x - 1) / block_size.x, (seq_len + block_size.y - 1) / block_size.y, batch_size * heads);
        mask_kernel<<<grid_size, block_size>>>(weight.data_gpu, batch_size, seq_len, heads);
        hipDeviceSynchronize();
        CHECK_ERROR();
        hipMemcpy(weight.data_cpu, weight.data_gpu, weight.size * sizeof(float), hipMemcpyDeviceToHost);
    }
    weight = weight * (1.0 / sqrt(d_head));
    weight = Tensor::softmax(weight);
    Tensor result = Tensor::matmul(weight, v).transpose(1, 2).reshape(input.dims, 3);
    return out_proj.forward(result);
}

void SelfAttention::free_memory() {
    q_proj.free_memory();
    k_proj.free_memory();
    v_proj.free_memory();
    out_proj.free_memory();
}

CrossAttention::CrossAttention() {
    q_proj = Linear();
    k_proj = Linear();
    v_proj = Linear();
    out_proj = Linear();
    heads = 0;
    d_embed = 0;
    d_head = 0;
}

CrossAttention::CrossAttention(int heads, int d_embed, int d_cross, bool proj_bias, bool out_bias) {
    q_proj = Linear(d_embed, d_embed, proj_bias);
    k_proj = Linear(d_cross, d_embed, proj_bias);
    v_proj = Linear(d_cross, d_embed, proj_bias);
    out_proj = Linear(d_embed, d_embed, out_bias);
    this->heads = heads;
    this->d_embed = d_embed;
    this->d_head = d_embed / heads;
    this->d_cross = d_cross;
}

Tensor CrossAttention::forward(Tensor& input, Tensor& context) {
    int batch_size = input.dims[0];
    int seq_len = input.dims[1];
    int interim_dims[] = {batch_size, seq_len, heads, d_head};
    Tensor q = q_proj.forward(input).reshape(interim_dims, 4).transpose(1, 2);
    Tensor k = k_proj.forward(context).reshape(interim_dims, 4).transpose(1, 2).transpose(2, 3);
    Tensor v = v_proj.forward(context).reshape(interim_dims, 4).transpose(1, 2);
    Tensor weight = Tensor::matmul(q, k);
    weight = weight * (1.0 / sqrt(d_head));
    weight = Tensor::softmax(weight);
    Tensor result = Tensor::matmul(weight, v).transpose(1, 2).reshape(input.dims, 3);
    return out_proj.forward(result);
}