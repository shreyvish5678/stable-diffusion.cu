#include "hip/hip_runtime.h"
#include "kernels.h"
#include "tensor.h"
#include "linear.h"
#include "attention.h"

Attention::Attention(int heads, int d_embed, bool proj_bias, bool out_bias) {
    q_proj = Linear(d_embed, d_embed, proj_bias);
    k_proj = Linear(d_embed, d_embed, proj_bias);
    v_proj = Linear(d_embed, d_embed, proj_bias);
    out_proj = Linear(d_embed, d_embed, out_bias);
    this->heads = heads;
    this->d_embed = d_embed;
    this->d_head = d_embed / heads;
}

Tensor Attention::forward(const Tensor& input, bool mask) {
    int batch_size = input.dims[0];
    int seq_len = input.dims[1];
    int interim_dims[] = {batch_size, seq_len, heads, d_embed};
    Tensor q = q_proj.forward(input).reshape(interim_dims, 4).transpose(1, 2);
    Tensor k = k_proj.forward(input).reshape(interim_dims, 4).transpose(1, 2);
    Tensor v = v_proj.forward(input).reshape(interim_dims, 4).transpose(1, 2);
    Tensor weight = Tensor::matmul(q, k.transpose(2, 3));
    if (mask) { 
        dim3 block_size(16, 16);
        dim3 grid_size(batch_size, seq_len, (heads + block_size.y - 1) / block_size.y);
        mask_kernel<<<grid_size, block_size>>>(weight.data_gpu, batch_size, seq_len, heads, d_head, 1);
    }
    weight = weight * (1.0 / sqrt(d_head));
    weight = weight.softmax();
    Tensor result = Tensor::matmul(weight, v).transpose(1, 2).reshape(input.dims, 3);
    return out_proj.forward(result);
}

int main() {
    Tensor input = Tensor(new int[3]{32, 128, 512}, 3);
    input.init_rand();
    return 0;
}